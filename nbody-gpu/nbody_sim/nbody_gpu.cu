#include <iostream>
#include <fstream>
#include <random>
#include <cmath>
#include <hip/hip_runtime.h>
#include <chrono>
#include <string>
#include <cstdlib>

double G = 6.674*std::pow(10,-11);
//double G = 1;

#define CUDA_CHECK(call) \
do { \
    hipError_t err = (call); \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA error at %s:%d - %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
        exit(EXIT_FAILURE); \
    } \
} while (0)

struct simulation {
  size_t nbpart;
  //change std::vector to pointers for cuda
  double* hmass;

  //host
  //position
  double* hx;
  double* hy;
  double* hz;

  //velocity
  double* hvx;
  double* hvy;
  double* hvz;

  //force
  double * hfx;
  double* hfy;
  double* hfz;

  //device
  double* dmass;
  double* dx;
  double* dy;
  double* dz;
  double* dvx;
  double* dvy;
  double* dvz;
  double* dfx;
  double* dfy;
  double* dfz;

  simulation(size_t nb) : nbpart(nb) {
    //host memory
    hmass = new double[nb]();
    hx = new double[nb](); 
    hy = new double[nb](); 
    hz = new double[nb]();
    hvx = new double[nb](); 
    hvy = new double[nb](); 
    hvz = new double[nb]();
    hfx = new double[nb](); 
    hfy = new double[nb](); 
    hfz = new double[nb]();

    //device memory
    CUDA_CHECK(hipMalloc(&dmass, nb * sizeof(double)));
    CUDA_CHECK(hipMalloc(&dx, nb * sizeof(double)));
    CUDA_CHECK(hipMalloc(&dy, nb * sizeof(double)));
    CUDA_CHECK(hipMalloc(&dz, nb * sizeof(double)));
    CUDA_CHECK(hipMalloc(&dvx, nb * sizeof(double)));
    CUDA_CHECK(hipMalloc(&dvy, nb * sizeof(double)));
    CUDA_CHECK(hipMalloc(&dvz, nb * sizeof(double)));
    CUDA_CHECK(hipMalloc(&dfx, nb * sizeof(double)));
    CUDA_CHECK(hipMalloc(&dfy, nb * sizeof(double)));
    CUDA_CHECK(hipMalloc(&dfz, nb * sizeof(double)));

    CUDA_CHECK(hipMemset(dmass, 0, nb * sizeof(double)));
    CUDA_CHECK(hipMemset(dx, 0, nb * sizeof(double)));
    CUDA_CHECK(hipMemset(dy, 0, nb * sizeof(double)));
    CUDA_CHECK(hipMemset(dz, 0, nb * sizeof(double)));
    CUDA_CHECK(hipMemset(dvx, 0, nb * sizeof(double)));
    CUDA_CHECK(hipMemset(dvy, 0, nb * sizeof(double)));
    CUDA_CHECK(hipMemset(dvz, 0, nb * sizeof(double)));
    CUDA_CHECK(hipMemset(dfx, 0, nb * sizeof(double)));
    CUDA_CHECK(hipMemset(dfy, 0, nb * sizeof(double)));
    CUDA_CHECK(hipMemset(dfz, 0, nb * sizeof(double)));
  }

  ~simulation() {
    //free memory
    delete[] hmass;
    delete[] hx; 
    delete[] hy; 
    delete[] hz;
    delete[] hvx; 
    delete[] hvy; 
    delete[] hvz;
    delete[] hfx; 
    delete[] hfy; 
    delete[] hfz;

    CUDA_CHECK(hipFree(dmass));
    CUDA_CHECK(hipFree(dx));
    CUDA_CHECK(hipFree(dy));
    CUDA_CHECK(hipFree(dz));
    CUDA_CHECK(hipFree(dvx));
    CUDA_CHECK(hipFree(dvy));
    CUDA_CHECK(hipFree(dvz));
    CUDA_CHECK(hipFree(dfx));
    CUDA_CHECK(hipFree(dfy));
    CUDA_CHECK(hipFree(dfz));
  }

  //resize function to handle changing particle count
  void resize(size_t new_nbpart) {
    if (new_nbpart == nbpart) return;

    this->~simulation();

    nbpart = new_nbpart;
    hmass = new double[nbpart]();
    hx = new double[nbpart](); 
    hy = new double[nbpart](); 
    hz = new double[nbpart]();
    hvx = new double[nbpart](); 
    hvy = new double[nbpart](); 
    hvz = new double[nbpart]();
    hfx = new double[nbpart](); 
    hfy = new double[nbpart](); 
    hfz = new double[nbpart]();

    CUDA_CHECK(hipMalloc(&dmass, nbpart * sizeof(double)));
    CUDA_CHECK(hipMalloc(&dx, nbpart * sizeof(double)));
    CUDA_CHECK(hipMalloc(&dy, nbpart * sizeof(double)));
    CUDA_CHECK(hipMalloc(&dz, nbpart * sizeof(double)));
    CUDA_CHECK(hipMalloc(&dvx, nbpart * sizeof(double)));
    CUDA_CHECK(hipMalloc(&dvy, nbpart * sizeof(double)));
    CUDA_CHECK(hipMalloc(&dvz, nbpart * sizeof(double)));
    CUDA_CHECK(hipMalloc(&dfx, nbpart * sizeof(double)));
    CUDA_CHECK(hipMalloc(&dfy, nbpart * sizeof(double)));
    CUDA_CHECK(hipMalloc(&dfz, nbpart * sizeof(double)));

    CUDA_CHECK(hipMemset(dmass, 0, nbpart * sizeof(double)));
    CUDA_CHECK(hipMemset(dx, 0, nbpart * sizeof(double)));
    CUDA_CHECK(hipMemset(dy, 0, nbpart * sizeof(double)));
    CUDA_CHECK(hipMemset(dz, 0, nbpart * sizeof(double)));
    CUDA_CHECK(hipMemset(dvx, 0, nbpart * sizeof(double)));
    CUDA_CHECK(hipMemset(dvy, 0, nbpart * sizeof(double)));
    CUDA_CHECK(hipMemset(dvz, 0, nbpart * sizeof(double)));
    CUDA_CHECK(hipMemset(dfx, 0, nbpart * sizeof(double)));
    CUDA_CHECK(hipMemset(dfy, 0, nbpart * sizeof(double)));
    CUDA_CHECK(hipMemset(dfz, 0, nbpart * sizeof(double)));
  }

  //copy from host to device
  void copy_to_device() {
    CUDA_CHECK(hipMemcpy(dmass, hmass, nbpart * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(dx, hx, nbpart * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(dy, hy, nbpart * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(dz, hz, nbpart * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(dvx, hvx, nbpart * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(dvy, hvy, nbpart * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(dvz, hvz, nbpart * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(dfx, hfx, nbpart * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(dfy, hfy, nbpart * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(dfz, hfz, nbpart * sizeof(double), hipMemcpyHostToDevice));
  }

  //copy from device to host
  void copy_from_device() {
    CUDA_CHECK(hipMemcpy(hx, dx, nbpart * sizeof(double), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(hy, dy, nbpart * sizeof(double), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(hz, dz, nbpart * sizeof(double), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(hvx, dvx, nbpart * sizeof(double), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(hvy, dvy, nbpart * sizeof(double), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(hvz, dvz, nbpart * sizeof(double), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(hfx, dfx, nbpart * sizeof(double), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(hfy, dfy, nbpart * sizeof(double), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(hfz, dfz, nbpart * sizeof(double), hipMemcpyDeviceToHost));
  }
};

void random_init(simulation& s) {
  std::random_device rd;  
  std::mt19937 gen(rd());
  std::uniform_real_distribution dismass(0.9, 1.);
  std::normal_distribution dispos(0., 1.);
  std::normal_distribution disvel(0., 1.);

  for (size_t i = 0; i<s.nbpart; ++i) {
    s.hmass[i] = dismass(gen);

    s.hx[i] = dispos(gen);
    s.hy[i] = dispos(gen);
    s.hz[i] = dispos(gen);
    s.hz[i] = 0.;
    
    s.hvx[i] = disvel(gen);
    s.hvy[i] = disvel(gen);
    s.hvz[i] = disvel(gen);
    s.hvz[i] = 0.;
    s.hvx[i] = s.hy[i]*1.5;
    s.hvy[i] = -s.hx[i]*1.5;

    s.hfx[i] = s.hfy[i] = s.hfz[i] = 0.0;
  }
  s.copy_to_device();
}

void init_solar(simulation& s) {
  if (s.nbpart != 10) {
    s.resize(10);
  }

  enum Planets {SUN, MERCURY, VENUS, EARTH, MARS, JUPITER, SATURN, URANUS, NEPTUNE, MOON};

  // Masses in kg
  s.hmass[SUN] = 1.9891 * std::pow(10, 30);
  s.hmass[MERCURY] = 3.285 * std::pow(10, 23);
  s.hmass[VENUS] = 4.867 * std::pow(10, 24);
  s.hmass[EARTH] = 5.972 * std::pow(10, 24);
  s.hmass[MARS] = 6.39 * std::pow(10, 23);
  s.hmass[JUPITER] = 1.898 * std::pow(10, 27);
  s.hmass[SATURN] = 5.683 * std::pow(10, 26);
  s.hmass[URANUS] = 8.681 * std::pow(10, 25);
  s.hmass[NEPTUNE] = 1.024 * std::pow(10, 26);
  s.hmass[MOON] = 7.342 * std::pow(10, 22);

  // Positions (in meters) and velocities (in m/s)
  double AU = 1.496 * std::pow(10, 11); // Astronomical Unit

  s.hx[SUN] = 0; s.hy[SUN] = 0; s.hz[SUN] = 0;
  s.hx[MERCURY] = 0.39*AU; s.hy[MERCURY] = 0; s.hz[MERCURY] = 0;
  s.hx[VENUS] = 0.72*AU; s.hy[VENUS] = 0; s.hz[VENUS] = 0;
  s.hx[EARTH] = 1.0*AU; s.hy[EARTH] = 0; s.hz[EARTH] = 0;
  s.hx[MARS] = 1.52*AU; s.hy[MARS] = 0; s.hz[MARS] = 0;
  s.hx[JUPITER] = 5.20*AU; s.hy[JUPITER] = 0; s.hz[JUPITER] = 0;
  s.hx[SATURN] = 9.58*AU; s.hy[SATURN] = 0; s.hz[SATURN] = 0;
  s.hx[URANUS] = 19.22*AU; s.hy[URANUS] = 0; s.hz[URANUS] = 0;
  s.hx[NEPTUNE] = 30.05*AU; s.hy[NEPTUNE] = 0; s.hz[NEPTUNE] = 0;
  s.hx[MOON] = 1.0*AU + 3.844*std::pow(10, 8); s.hy[MOON] = 0; s.hz[MOON] = 0;

  s.hvx[SUN] = 0; s.hvy[SUN] = 0; s.hvz[SUN] = 0;
  s.hvx[MERCURY] = 0; s.hvy[MERCURY] = 47870; s.hvz[MERCURY] = 0;
  s.hvx[VENUS] = 0; s.hvy[VENUS] = 35020; s.hvz[VENUS] = 0;
  s.hvx[EARTH] = 0; s.hvy[EARTH] = 29780; s.hvz[EARTH] = 0;
  s.hvx[MARS] = 0; s.hvy[MARS] = 24130; s.hvz[MARS] = 0;
  s.hvx[JUPITER] = 0; s.hvy[JUPITER] = 13070; s.hvz[JUPITER] = 0;
  s.hvx[SATURN] = 0; s.hvy[SATURN] = 9680; s.hvz[SATURN] = 0;
  s.hvx[URANUS] = 0; s.hvy[URANUS] = 6800; s.hvz[URANUS] = 0;
  s.hvx[NEPTUNE] = 0; s.hvy[NEPTUNE] = 5430; s.hvz[NEPTUNE] = 0;
  s.hvx[MOON] = 0; s.hvy[MOON] = 29780 + 1022; s.hvz[MOON] = 0;

  for (int i = 0; i < 10; i++) {
      s.hfx[i] = 0;
      s.hfy[i] = 0;
      s.hfz[i] = 0;
  }
  s.copy_to_device();
}

//cuda kernel for computing forces
__global__ void compute_force_kernel(double* mass, double* x, double* y, double* z, double* fx, double* fy, double* fz, size_t nbpart, double G) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= nbpart) return;

  double softening = 0.1;
  double my_x = x[i];
  double my_y = y[i];
  double my_z = z[i];
  double my_fx = 0.0;
  double my_fy = 0.0;
  double my_fz = 0.0;

  for (int j = 0; j < nbpart; j++) {
  if (i == j) continue;

  double dx = x[j] - my_x;
  double dy = y[j] - my_y;
  double dz = z[j] - my_z;

  double dist_sq = dx*dx + dy*dy + dz*dz + softening;
  double inv_dist = rsqrt(dist_sq);
  double inv_dist3 = inv_dist * inv_dist * inv_dist;

  double F = G * mass[i] * mass[j] * inv_dist3;

  my_fx += F * dx;
  my_fy += F * dy;
  my_fz += F * dz;
  }

  fx[i] = my_fx;
  fy[i] = my_fy;
  fz[i] = my_fz;
}

//cuda kernel for updating positions
__global__ void update_particles_kernel(double* x, double* y, double* z, double* vx, double* vy, double* vz, double* fx, double* fy, double* fz, double* mass, size_t nbpart, double dt) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= nbpart) return;

  // Update velocity
  vx[i] += fx[i] / mass[i] * dt;
  vy[i] += fy[i] / mass[i] * dt;
  vz[i] += fz[i] / mass[i] * dt;

  // Update position
  x[i] += vx[i] * dt;
  y[i] += vy[i] * dt;
  z[i] += vz[i] * dt;
}

//cuda kernel for resetting forces
__global__ void reset_force_kernel(double* fx, double* fy, double* fz, size_t nbpart) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < nbpart) {
    fx[i] = 0.0;
    fy[i] = 0.0;
    fz[i] = 0.0;
  }
}

void dump_state(simulation& s) {
  std::cout<<s.nbpart<<'\t';
  for (size_t i=0; i<s.nbpart; ++i) {
    std::cout<<s.hmass[i]<<'\t';
    std::cout<<s.hx[i]<<'\t'<<s.hy[i]<<'\t'<<s.hz[i]<<'\t';
    std::cout<<s.hvx[i]<<'\t'<<s.hvy[i]<<'\t'<<s.hvz[i]<<'\t';
    std::cout<<s.hfx[i]<<'\t'<<s.hfy[i]<<'\t'<<s.hfz[i]<<'\t';
  }
  std::cout<<'\n';
}

void loadfrom_file(simulation& s, std::string filename) {
  std::ifstream in(filename);
  if (!in.is_open()) {
    std::cerr << "ERROR: COULD NOT OPEN FILE " << filename << std::endl;
    exit(EXIT_FAILURE);
  }
  size_t nbpart;
  in >> nbpart;
  if (s.nbpart != nbpart) {
    s.resize(nbpart);
  }
  for (size_t i=0; i<s.nbpart; ++i) {
    in >> s.hmass[i];
    in >> s.hx[i] >> s.hy[i] >> s.hz[i];
    in >> s.hvx[i] >> s.hvy[i] >> s.hvz[i];
    in >> s.hfx[i] >> s.hfy[i] >> s.hfz[i];
  }
  if (!in.good())
    throw "kaboom";
  s.copy_to_device();
}

int main(int argc, char* argv[]) {
  if (argc != 6) {
    std::cerr
      <<"usage: "<<argv[0]<<" <input> <dt> <nbstep> <printevery> <blocksize>"<<"\n"
      <<"input can be:"<<"\n"
      <<"a number (random initialization)"<<"\n"
      <<"planet (initialize with solar system)"<<"\n"
      <<"a filename (load from file in singleline tsv)"<<"\n";
    return -1;
  }
  
  double dt = std::atof(argv[2]); //in seconds
  size_t nbstep = std::atol(argv[3]);
  size_t printevery = std::atol(argv[4]);
  int blockSize = std::atol(argv[5]);
  
  
  simulation s(1);

  //parse command line
  {
    size_t nbpart = std::atol(argv[1]); //return 0 if not a number
    if ( nbpart > 0) {
      if (s.nbpart != nbpart) {
        s.resize(nbpart);
      }
      random_init(s);
    } else {
      std::string inputparam = argv[1];
      if (inputparam == "planet") {
        if (s.nbpart != 10) {
          s.resize(10);
        }
        init_solar(s); 
      } else{
	loadfrom_file(s, inputparam);
      }
    }    
  }

  int numBlocks = (s.nbpart + blockSize - 1) / blockSize;

  auto start = std::chrono::high_resolution_clock::now();
  for (size_t step = 0; step< nbstep; step++) {
    /*if (step %printevery == 0) {
      s.copy_from_device();
    }*/
  reset_force_kernel<<<numBlocks, blockSize>>>(s.dfx, s.dfy, s.dfz, s.nbpart);
  compute_force_kernel<<<numBlocks, blockSize>>>(s.dmass, s.dx, s.dy, s.dz, s.dfx, s.dfy, s.dfz, s.nbpart, G);
  update_particles_kernel<<<numBlocks, blockSize>>>(s.dx, s.dy, s.dz, s.dvx, s.dvy, s.dvz, s.dfx, s.dfy, s.dfz, s.dmass, s.nbpart, dt);
  }
  CUDA_CHECK(hipDeviceSynchronize());
  CUDA_CHECK(hipGetLastError());

  auto end = std::chrono::high_resolution_clock::now();
  std::chrono::duration<double> elapsed = end - start;
  std::cout << "GPU Time: " << elapsed.count() << " s" << std::endl;
  
  //s.copy_from_device();
  //dump_state(s);  


  return 0;
}
