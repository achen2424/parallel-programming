#include <iostream>
#include <fstream>
#include <random>
#include <cmath>
#include <hip/hip_runtime.h>
#include <chrono>

double G = 6.674*std::pow(10,-11);
//double G = 1;

struct simulation {
  size_t nbpart;
  //change std::vector to pointers for cuda
  double* hmass;

  //host
  //position
  double* hx;
  double* hy;
  double* hz;

  //velocity
  double* hvx;
  double* hvy;
  double* hvz;

  //force
  double * hfx;
  double* hfy;
  double* hfz;

  //device
  double* dmass;
  double* dx;
  double* dy;
  double* dz;
  double* dvx;
  double* dvy;
  double* dvz;
  double* dfx;
  double* dfy;
  double* dfz;

  simulation(size_t nb) : nbpart(nb) {
    //host memory
    hmass = new double[nb];
    hx = new double[nb]; 
    hy = new double[nb]; 
    hz = new double[nb];
    hvx = new double[nb]; 
    hvy = new double[nb]; 
    hvz = new double[nb];
    hfx = new double[nb]; 
    hfy = new double[nb]; 
    hfz = new double[nb];

    //device memory
    hipMalloc(&dmass, nb * sizeof(double));
    hipMalloc(&dx, nb * sizeof(double));
    hipMalloc(&dy, nb * sizeof(double));
    hipMalloc(&dz, nb * sizeof(double));
    hipMalloc(&dvx, nb * sizeof(double));
    hipMalloc(&dvy, nb * sizeof(double));
    hipMalloc(&dvz, nb * sizeof(double));
    hipMalloc(&dfx, nb * sizeof(double));
    hipMalloc(&dfy, nb * sizeof(double));
    hipMalloc(&dfz, nb * sizeof(double));
  }

  ~simulation() {
    //free memory
    delete[] hmass;
    delete[] hx; 
    delete[] hy; 
    delete[] hz;
    delete[] hvx; 
    delete[] hvy; 
    delete[] hvz;
    delete[] hfx; 
    delete[] hfy; 
    delete[] hfz;

    hipFree(dmass);
    hipFree(dx);
    hipFree(dy);
    hipFree(dz);
    hipFree(dvx);
    hipFree(dvy);
    hipFree(dvz);
    hipFree(dfx);
    hipFree(dfy);
    hipFree(dfz);
  }

  //copy from host to device
  void copy_to_device() {
    hipMemcpy(dmass, hmass, nbpart * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dx, hx, nbpart * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dy, hy, nbpart * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dz, hz, nbpart * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dvx, hvx, nbpart * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dvy, hvy, nbpart * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dvz, hvz, nbpart * sizeof(double), hipMemcpyHostToDevice);
  }

  //copy from device to host
  void copy_from_device() {
    hipMemcpy(hx, dx, nbpart * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(hy, dy, nbpart * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(hz, dz, nbpart * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(hvx, dvx, nbpart * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(hvy, dvy, nbpart * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(hvz, dvz, nbpart * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(hfx, dfx, nbpart * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(hfy, dfy, nbpart * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(hfz, dfz, nbpart * sizeof(double), hipMemcpyDeviceToHost);
  }
};

void random_init(simulation& s) {
  std::random_device rd;  
  std::mt19937 gen(rd());
  std::uniform_real_distribution dismass(0.9, 1.);
  std::normal_distribution dispos(0., 1.);
  std::normal_distribution disvel(0., 1.);

  for (size_t i = 0; i<s.nbpart; ++i) {
    s.hmass[i] = dismass(gen);

    s.hx[i] = dispos(gen);
    s.hy[i] = dispos(gen);
    s.hz[i] = dispos(gen);
    s.hz[i] = 0.;
    
    s.hvx[i] = disvel(gen);
    s.hvy[i] = disvel(gen);
    s.hvz[i] = disvel(gen);
    s.hvz[i] = 0.;
    s.hvx[i] = s.hy[i]*1.5;
    s.hvy[i] = -s.hx[i]*1.5;
  }
  s.copy_to_device();
}

void init_solar(simulation& s) {
  enum Planets {SUN, MERCURY, VENUS, EARTH, MARS, JUPITER, SATURN, URANUS, NEPTUNE, MOON};
  s = simulation(10);

  // Masses in kg
  s.hmass[SUN] = 1.9891 * std::pow(10, 30);
  s.hmass[MERCURY] = 3.285 * std::pow(10, 23);
  s.hmass[VENUS] = 4.867 * std::pow(10, 24);
  s.hmass[EARTH] = 5.972 * std::pow(10, 24);
  s.hmass[MARS] = 6.39 * std::pow(10, 23);
  s.hmass[JUPITER] = 1.898 * std::pow(10, 27);
  s.hmass[SATURN] = 5.683 * std::pow(10, 26);
  s.hmass[URANUS] = 8.681 * std::pow(10, 25);
  s.hmass[NEPTUNE] = 1.024 * std::pow(10, 26);
  s.hmass[MOON] = 7.342 * std::pow(10, 22);

  // Positions (in meters) and velocities (in m/s)
  double AU = 1.496 * std::pow(10, 11); // Astronomical Unit

  s.hx[SUN] = 0; s.hy[SUN] = 0; s.hz[SUN] = 0;
  s.hx[MERCURY] = 0.39*AU; s.hy[MERCURY] = 0; s.hz[MERCURY] = 0;
  s.hx[VENUS] = 0.72*AU; s.hy[VENUS] = 0; s.hz[VENUS] = 0;
  s.hx[EARTH] = 1.0*AU; s.hy[EARTH] = 0; s.hz[EARTH] = 0;
  s.hx[MARS] = 1.52*AU; s.hy[MARS] = 0; s.hz[MARS] = 0;
  s.hx[JUPITER] = 5.20*AU; s.hy[JUPITER] = 0; s.hz[JUPITER] = 0;
  s.hx[SATURN] = 9.58*AU; s.hy[SATURN] = 0; s.hz[SATURN] = 0;
  s.hx[URANUS] = 19.22*AU; s.hy[URANUS] = 0; s.hz[URANUS] = 0;
  s.hx[NEPTUNE] = 30.05*AU; s.hy[NEPTUNE] = 0; s.hz[NEPTUNE] = 0;
  s.hx[MOON] = 1.0*AU + 3.844*std::pow(10, 8); s.hy[MOON] = 0; s.hz[MOON] = 0;

  s.hvx[SUN] = 0; s.hvy[SUN] = 0; s.hvz[SUN] = 0;
  s.hvx[MERCURY] = 0; s.hvy[MERCURY] = 47870; s.hvz[MERCURY] = 0;
  s.hvx[VENUS] = 0; s.hvy[VENUS] = 35020; s.hvz[VENUS] = 0;
  s.hvx[EARTH] = 0; s.hvy[EARTH] = 29780; s.hvz[EARTH] = 0;
  s.hvx[MARS] = 0; s.hvy[MARS] = 24130; s.hvz[MARS] = 0;
  s.hvx[JUPITER] = 0; s.hvy[JUPITER] = 13070; s.hvz[JUPITER] = 0;
  s.hvx[SATURN] = 0; s.hvy[SATURN] = 9680; s.hvz[SATURN] = 0;
  s.hvx[URANUS] = 0; s.hvy[URANUS] = 6800; s.hvz[URANUS] = 0;
  s.hvx[NEPTUNE] = 0; s.hvy[NEPTUNE] = 5430; s.hvz[NEPTUNE] = 0;
  s.hvx[MOON] = 0; s.hvy[MOON] = 29780 + 1022; s.hvz[MOON] = 0;

  for (int i = 0; i < 10; i++) {
      s.hfx[i] = 0;
      s.hfy[i] = 0;
      s.hfz[i] = 0;
  }
  s.copy_to_device();
}

//cuda kernel for computing forces
__global__ void compute_force_kernel(double* mass, double* x, double* y, double* z, double* fx, double* fy, double* fz, size_t nbpart, double G) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= nbpart) return;

  double softening = 0.1;
  double my_x = x[i];
  double my_y = y[i];
  double my_z = z[i];
  double my_fx = 0.0;
  double my_fy = 0.0;
  double my_fz = 0.0;

  for (int j = 0; j < nbpart; j++) {
  if (i == j) continue;

  double dx = x[j] - my_x;
  double dy = y[j] - my_y;
  double dz = z[j] - my_z;

  double dist_sq = dx*dx + dy*dy + dz*dz + softening;
  double inv_dist = rsqrt(dist_sq);
  double inv_dist3 = inv_dist * inv_dist * inv_dist;

  double F = G * mass[i] * mass[j] * inv_dist3;

  my_fx += F * dx;
  my_fy += F * dy;
  my_fz += F * dz;
  }

  fx[i] = my_fx;
  fy[i] = my_fy;
  fz[i] = my_fz;
}

//cuda kernel for updating positions
__global__ void update_particles_kernel(double* x, double* y, double* z, double* vx, double* vy, double* vz, double* fx, double* fy, double* fz, double* mass, size_t nbpart, double dt) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= nbpart) return;

  // Update velocity
  vx[i] += fx[i] / mass[i] * dt;
  vy[i] += fy[i] / mass[i] * dt;
  vz[i] += fz[i] / mass[i] * dt;

  // Update position
  x[i] += vx[i] * dt;
  y[i] += vy[i] * dt;
  z[i] += vz[i] * dt;
}

//cuda kernel for resetting forces
__global__ void reset_force_kernel(double* fx, double* fy, double* fz, size_t nbpart) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < nbpart) {
    fx[i] = 0.0;
    fy[i] = 0.0;
    fz[i] = 0.0;
  }
}

void dump_state(simulation& s) {
  std::cout<<s.nbpart<<'\t';
  for (size_t i=0; i<s.nbpart; ++i) {
    std::cout<<s.hmass[i]<<'\t';
    std::cout<<s.hx[i]<<'\t'<<s.hy[i]<<'\t'<<s.hz[i]<<'\t';
    std::cout<<s.hvx[i]<<'\t'<<s.hvy[i]<<'\t'<<s.hvz[i]<<'\t';
    std::cout<<s.hfx[i]<<'\t'<<s.hfy[i]<<'\t'<<s.hfz[i]<<'\t';
  }
  std::cout<<'\n';
}

void loadfrom_file(simulation& s, std::string filename) {
  std::ifstream in (filename);
  size_t nbpart;
  in>>nbpart;
  s = simulation(nbpart);
  for (size_t i=0; i<s.nbpart; ++i) {
    in>>s.hmass[i];
    in >>  s.hx[i] >>  s.hy[i] >>  s.hz[i];
    in >> s.hvx[i] >> s.hvy[i] >> s.hvz[i];
    in >> s.hfx[i] >> s.hfy[i] >> s.hfz[i];
  }
  if (!in.good())
    throw "kaboom";
  s.copy_to_device();
}

int main(int argc, char* argv[]) {
  if (argc != 6) {
    std::cerr
      <<"usage: "<<argv[0]<<" <input> <dt> <nbstep> <printevery> <blocksize>"<<"\n"
      <<"input can be:"<<"\n"
      <<"a number (random initialization)"<<"\n"
      <<"planet (initialize with solar system)"<<"\n"
      <<"a filename (load from file in singleline tsv)"<<"\n";
    return -1;
  }
  
  double dt = std::atof(argv[2]); //in seconds
  size_t nbstep = std::atol(argv[3]);
  size_t printevery = std::atol(argv[4]);
  int blockSize = std::atol(argv[5]);
  
  
  simulation s(1);

  //parse command line
  {
    size_t nbpart = std::atol(argv[1]); //return 0 if not a number
    if ( nbpart > 0) {
      s = simulation(nbpart);
      random_init(s);
    } else {
      std::string inputparam = argv[1];
      if (inputparam == "planet") {
	init_solar(s);
      } else{
	loadfrom_file(s, inputparam);
      }
    }    
  }

  int numBlocks = (s.nbpart + blockSize - 1) / blockSize;

  auto start = std::chrono::high_resolution_clock::now();
  for (size_t step = 0; step< nbstep; step++) {
    /*if (step %printevery == 0) {
      s.copy_from_device();
    }*/
  reset_force_kernel<<<numBlocks, blockSize>>>(s.dfx, s.dfy, s.dfz, s.nbpart);
  compute_force_kernel<<<numBlocks, blockSize>>>(s.dmass, s.dx, s.dy, s.dz, s.dfx, s.dfy, s.dfz, s.nbpart, G);
  update_particles_kernel<<<numBlocks, blockSize>>>(s.dx, s.dy, s.dz, s.dvx, s.dvy, s.dvz, s.dfx, s.dfy, s.dfz, s.dmass, s.nbpart, dt);
  }
  hipDeviceSynchronize();
  auto end = std::chrono::high_resolution_clock::now();
  std::chrono::duration<double> elapsed = end - start;
  std::cout << "GPU Time: " << elapsed.count() << " s\n";
  
  //s.copy_from_device();
  //dump_state(s);  


  return 0;
}
